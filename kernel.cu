#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cmath>
#include <cstdlib>

const int WIDTH = 3840;
const int HEIGHT = 2160;
const int MAX_ITER = 1000;
const int TOTAL_FRAMES = 900;  // Total number of frames
const double INITIAL_ZOOM_FACTOR = 0.9975;  // Initial zoom factor per frame

__global__ void mandelbrot_kernel(unsigned char* output, double center_x, double center_y, double scale, double real_min, double real_max, double imag_min, double imag_max)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= WIDTH || y >= HEIGHT) return;

    double real = real_min + (real_max - real_min) * x / WIDTH;
    double imag = imag_min + (imag_max - imag_min) * y / HEIGHT;

    double c_re = real;
    double c_im = imag;
    int iter = 0;

    while (real * real + imag * imag <= 4.0 && iter < MAX_ITER)
    {
        double real_new = real * real - imag * imag + c_re;
        double imag_new = 2.0 * real * imag + c_im;
        real = real_new;
        imag = imag_new;
        iter++;
    }

    // Simplified coloring without smooth iteration
    unsigned char r = (unsigned char)(iter * 1) % 256;
    unsigned char g = (unsigned char)(iter * 2) % 256;
    unsigned char b = (unsigned char)(iter * 4) % 256;

    int idx = (y * WIDTH + x) * 3;

    output[idx + 0] = r;
    output[idx + 1] = g;
    output[idx + 2] = b;
}

void save_ppm(const char* filename, unsigned char* data)
{
    FILE* f = fopen(filename, "wb");
    if (!f) {
        printf("Failed to open file for writing: %s\n", filename);
        return;
    }
    fprintf(f, "P6\n%d %d\n255\n", WIDTH, HEIGHT);
    fwrite(data, 1, WIDTH * HEIGHT * 3, f);
    fclose(f);
}

int main()
{
    unsigned char* d_output, * h_output;
    size_t size = WIDTH * HEIGHT * 3 * sizeof(unsigned char);

    hipMalloc(&d_output, size);
    h_output = (unsigned char*)malloc(size);

    dim3 block(32, 32);
    dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);

    // Starting view parameters
    double center_x = 0.0;  // Start from the center of the Mandelbrot set
    double center_y = 0.0;  // Start from the center of the Mandelbrot set
    double scale = 2.0;      // Large scale to show the entire Mandelbrot set

    // The target coordinates to zoom into
    double target_x = -0.10944534372538328;
    double target_y = -0.8948242213462949;

    printf("Starting zoom sequence...\n");

    // Zoom into the target coordinates over multiple frames
    for (int frame = 0; frame < TOTAL_FRAMES; ++frame)
    {
        printf("Rendering frame %d/%d...\n", frame + 1, TOTAL_FRAMES);

        // Calculate the scale and pan (zoom) towards the target point
        double zoom_factor = INITIAL_ZOOM_FACTOR;

        // Gradually zoom in towards the target
        center_x += (target_x - center_x) * zoom_factor; // Pan to target_x
        center_y += (target_y - center_y) * zoom_factor; // Pan to target_y
        scale *= zoom_factor; // Decrease the scale for zoom

        // Precompute constants for real_min, real_max, imag_min, imag_max
        double real_min = center_x - scale;
        double real_max = center_x + scale;
        double imag_min = center_y - scale * HEIGHT / WIDTH;
        double imag_max = center_y + scale * HEIGHT / WIDTH;

        // Launch kernel with the current zoom parameters
        mandelbrot_kernel << <grid, block >> > (d_output, center_x, center_y, scale, real_min, real_max, imag_min, imag_max);

        // Copy the result from device to host
        hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

        // Save current frame to PPM file
        char filename[256];
        sprintf(filename, "C:\\users\\omere\\desktop\\img\\frame%04d.ppm", frame);
        save_ppm(filename, h_output);
    }

    hipFree(d_output);
    free(h_output);

    printf("All frames rendered!\n");
    system("pause");
    return 0;
}
